#include "hip/hip_runtime.h"
/*

Copyright 2016 Thomas Luu

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.

*/

/*

File: sncdfinv.cu

Computation of the skew-normal quantile function.

Based on:

Luu, T; (2016) Fast and accurate parallel computation of quantile functions for 
random number generation. Doctoral thesis, UCL (University College London).

http://discovery.ucl.ac.uk/1482128/

*/

#ifndef SNCDFINV
#define SNCDFINV

#include "plog.cu"

#include <hip/hip_math_constants.h>

__host__ __device__ double sncdfinv(double u, double a)
{
  double tol = 0.01;

  if (u == 0.0) {
#ifdef __CUDA_ARCH__
    return -HIP_INF;
#else
    return -INFINITY;
#endif
  }

  if (u == 1.0) {
#ifdef __CUDA_ARCH__
    return HIP_INF;
#else
    return INFINITY;
#endif
  }

  /*
   * Change of variable + special cases
   */
  if (a == 1) u = sqrt(u);
  if (a == -1) u = sqrt(1-u);
  double z = normcdfinv(u);
  if (a == 0) return z;
  else if (a == 1) return z;
  else if (a == -1) return -z;
  if (a < 0) z = -z;

  double A = fabs(a);

  double right_limit = erf(erfcinv(2 * tol) / A);

  /*
   * Tails
   */
  if (a > 0 && u > right_limit) {
    return 1.4142135623730950488 * erfinv(u);
  } else if (a < 0 && (1-u) > right_limit) {
    return -1.4142135623730950488 * erfcinv(u);
  }

  double x = normcdfinv(0.5 - 0.31830988618379067154 * atan(A));

  double expon = exp(0.5 * ( - x*x));
  double errfn = 1.0;
  double efder = expon * 0.79788456080286535588 * A / errfn;

  double c0 = 0;
  double c1 = expon / errfn;
  double c2 = - expon*(efder + errfn*x) / (2*errfn*errfn);
  double c3 = 0.16666666666666666667 * expon*(3*efder*efder + errfn*errfn*(-1 + x*x) + expon*expon + efder*(3*errfn*x)) / (errfn*errfn*errfn);
  double c4 = - 0.041666666666666666667 * expon*(15*efder*efder*efder + errfn*errfn*errfn*x*(-3 + x*x) + 6*errfn*expon*expon*x + efder*efder*(18*errfn*x) + efder*(errfn*errfn*(-4 + 7*x*x) + expon*expon*(7 - A*A))) / (errfn*errfn*errfn*errfn);
  double c5 = 0.0083333333333333333333 * expon*(105*efder*efder*efder*efder + errfn*errfn*errfn*errfn*(3 - 6*x*x + x*x*x*x) + 5*errfn*errfn*expon*expon*(-2 + 5*x*x) + expon*expon*expon*expon*(7) + 15*efder*efder*efder*(10*errfn*x) + efder*(5*errfn*errfn*errfn*x*(-5 + 3*x*x) + 10*errfn*expon*expon*x*(7 - A*A)) + 5*efder*efder*(3*errfn*errfn*(-2 + 5*x*x) + expon*expon*(-3*(-4 + A*A)))) / (errfn*errfn*errfn*errfn*errfn);

  //double h = 0.5 * pow(fabs(tol / c5), 0.2);
  double h = 0.75 * pow(fabs(tol / c5), 0.2);
  //double h = 0.9 * pow(fabs(tol / c5), 0.2);

  double left_limit = x - h;
  if (z < left_limit) {
    if (a > 0) {
      return -sqrt(2 * plog(1 / (6.2831853071795864769 * u * a)) / (1 + a*a));
    } else {
      return sqrt(2 * plog(1 / (6.2831853071795864769 * (1-u) * fabs(a))) / (1 + a*a));
    }
  }

  // otherwise eval central series 
  h = z - x;
  double res = c0 + h*(c1 + h*(c2 + h*(c3 + h*(c4 + h*c5))));

  return a < 0 ? -res : res;
}

#endif
